#include "hip/hip_runtime.h"
/*
����˷���CPU�汾��GPU�汾�ĶԱ�


*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <Windows.h>
#include <string>
#include <malloc.h>

//����ָʾ��ͬ��GPU �Ż��汾
enum Type
{
	Mode1 = 1,   //Mode 1 :��ÿһ��C[i][j]���ֱ����һ���߳�
	Mode2 = 2	 //Mode 2 :����һ���߳���������һ��C[i][j]��ͨ��C(i,j) = sum { A(i,k)*B(k,j) }���֣����ǻ�������ϸ�Ȼ��֣�
	//		   sub(i,j) = sum{A(i,ksub+offsetA)*B(ksub+offsetB,j)}  0<=ksub < blockSize
	//			C(i, j) = sum{ Csub(i, j) }
	//			���ǰѾ���ֳ�n*n������ӿ飬Ȼ��ÿһ��block��������ӿ�i �� �ӿ�j���ӳ˻���
	//			������Ϻ��������ɡ�������Ҫʹ���˹����Դ����Ż���
	//			����һ�� shared memory�ķ�����
};

texture<float, 2, hipReadModeElementType> texA;
texture<float, 2, hipReadModeElementType> texB;

__global__ void MatrixMul(float *c, unsigned int w, unsigned int h)
{
	float sum = 0;
	//�ҳ����߳����ڵ��к���
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	/*
	//������������
	float u = row / (float)w;
	float v = col / (float)h;

	//�߳�Thread(row, col)�������C(row, col)
	u -= 0.5f;
	v -= 0.5f;
	*/

	for (int i = 0; i < w; ++i)
	{
		sum += tex2D(texA, i, row) * tex2D(texB, col, i);
	}

	c[row * w + col] = sum;
}

hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int WA, unsigned int HA, unsigned int WB, unsigned int HB, Type mode);

__global__ void MatrixMulGPU_1(float *c, const float *a, const float *b, unsigned int WA, unsigned int WB)
{
	float sum = 0;
	//�ҳ����߳����ڵ��к���
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	//�߳�Thread(row, col)�������C(row, col)
	for (int i = 0; i < WB; ++i)
	{
		sum += a[row * WA + i] * b[i * WB + col];
	}

	c[row * WB + col] = sum;
}

template<int BLOCK_SIZE> __global__ void MatrixMulGPU_2(float *c, const float *a, const float *b, unsigned int WA, unsigned int WB)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = WA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + WA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B 
	int bStep = BLOCK_SIZE * WB;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int i = aBegin, j = bBegin;
		i <= aEnd;
		i += aStep, j += bStep)
	{

		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = a[i + WA * ty + tx];
		Bs[ty][tx] = b[j + WB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int k = WB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	c[k + WB * ty + tx] = Csub;
}

//GPU version
void MatrixMulCPU(float *_C, const float* _A, const float* _B, int WA, int HA, int WB, int HB)
{
	if (WA != HB)
	{
		printf("the matrix A and B cannot be multipled!");
		exit(0);
	}

	for (int i = 0; i < HA; ++i)
	{
		for (int j = 0; j < WB; ++j)
		{
			for (int k = 0; k < WA; ++k)
			{
				_C[i * WA + j] += _A[i * WA + k] * _B[k * WB + j];
			}
		}
	}
}

//����ʼ�ľ���һ�����ֵ
void randomInit(float* _data, int _size)
{
	for (int i = 0; i < _size; ++i)
	{
		_data[i] = rand() / (float)RAND_MAX;
	}
}

//print the matrix
void printMatrix(float* m_Matrix, int W, int H)
{
	for (int i = 0; i < W * H; ++i)
	{
		printf("%2.1f ", m_Matrix[i]);
		if (i % W == 0 && i != 0) printf("\n");
	}
	printf("\n");
}

bool CheckAnswer(const float* _C, const float* _D, unsigned int size)
{
	bool isRight = true;
	for (int i = 0; i < size /*&& isRight == true*/; ++i)
	{
		if (_C[i] != _D[i])
		{
			isRight = false;
			printf("%d : %3.8f != %3.8f \n", i, _C[i], _D[i]);
		}
			
	}

	return isRight;
}

int main()
{
	const int width_A = 1024;
	const int height_A = 1024;
	const int width_B = 1024;
	const int height_B = 1024;

	//����CPU�е�һ���ڴ�
	float *B = (float *)malloc(sizeof(float) * height_B * width_B);
	float *A = (float *)malloc(sizeof(float) * height_A * width_A);
	float *C = (float *)malloc(sizeof(float) * height_A * width_B);
	float *D = (float *)malloc(sizeof(float) * height_A * width_B);
	float *E = (float *)malloc(sizeof(float) * height_A * width_B);
	float *F = (float *)malloc(sizeof(float) * height_A * width_B);

	//��ʼ������ڴ�
	memset(A, 0.0, sizeof(float) * height_A * width_A);
	memset(B, 0.0, sizeof(float) * height_B * width_B);
	memset(C, 0.0, sizeof(float) * height_A * width_B);
	memset(D, 0.0, sizeof(float) * height_A * width_B);
	memset(E, 0.0, sizeof(float) * height_A * width_B);
	memset(F, 0.0, sizeof(float) * height_A * width_B);

	Type m_Mode;

	//���������������
	srand((unsigned)time(0));

	printf("������ɾ���A <height : %d, width : %d> ...\n", height_A, width_A);
	randomInit(B, height_B * width_B);
	printf("������ɾ���B <height : %d, width : %d> ...\n", height_B, width_B);
	randomInit(A, height_A * width_A);

	//printMatrix(B, width_B, height_B);
	//printMatrix(A, width_A, height_A);

	//CPU ����
	printf("CPU matrix multiplication...\n");
	unsigned int tick1 = GetTickCount();
	MatrixMulCPU(C, A, B, width_A, height_A, width_B, height_B);
	printf("CPU use time : %dms\n", GetTickCount() - tick1);


		//GPU 
		printf("GPU  normal matrix multiplication...\n");
		m_Mode = Mode1;

		//���뵽GPU
		hipError_t cudaStatus = addWithCuda(D, A, B, width_A, height_A, width_B, height_B, m_Mode);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "addWithCuda failed!\n");
			return 1;
		}

		printf("GPU  with shared memory and matrix being blocked matrix multiplication...\n");
		m_Mode = Mode2;

		cudaStatus = addWithCuda(E, A, B, width_A, height_A, width_B, height_B, m_Mode);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "addWithCuda failed!\n");
			return 1;
		}
	
		//GPU mode3 with texture memory

		hipChannelFormatDesc channelDescA = hipCreateChannelDesc((int)sizeof(float) * 8, 0, 0, 0, hipChannelFormatKindFloat);
		hipChannelFormatDesc channelDescB = hipCreateChannelDesc((int)sizeof(float) * 8, 0, 0, 0, hipChannelFormatKindFloat);

		hipArray* mat_A;
		hipArray* mat_B;

		hipMallocArray(&mat_A, &channelDescA, width_A, height_A);
		hipMallocArray(&mat_B, &channelDescB, width_B, height_B);

		hipMemcpyToArray(mat_A, 0, 0, A, sizeof(float) * height_A * width_A, hipMemcpyHostToDevice);
		hipMemcpyToArray(mat_B, 0, 0, B, sizeof(float) * height_B * width_B, hipMemcpyHostToDevice);

		//texA.addressMode[0] = hipAddressModeWrap;
		//texA.addressMode[1] = hipAddressModeWrap;
		texA.filterMode = hipFilterModePoint;
		texA.normalized = false;
		//texB.addressMode[0] = hipAddressModeWrap;
		//texB.addressMode[1] = hipAddressModeWrap;
		texB.filterMode = hipFilterModePoint;
		texB.normalized = false;

		hipBindTextureToArray(texA, mat_A, channelDescA);
		hipBindTextureToArray(texB, mat_B, channelDescB);

		float* d_C = NULL;
		hipMalloc(&d_C, width_B * height_A * sizeof(float));

		int block_size = 32;

		dim3 Threads(block_size, block_size);
		dim3 Blocks(width_B / block_size, height_A / block_size);

		hipEvent_t start3, stop3;
		hipEventCreate(&start3);
		hipEventCreate(&stop3);
		hipEventRecord(start3, 0);

		MatrixMul << < Threads, Blocks >> >(d_C, width_B, height_A);

		hipEventRecord(stop3, 0);
		hipEventSynchronize(stop3);

		float elaspsedTime3;
		hipEventElapsedTime(&elaspsedTime3, start3, stop3);
		printf("GPU with Texutre Memory time : %3.1fms \n", elaspsedTime3);
		hipMemcpy(F, d_C, sizeof(float) * width_B * height_A, hipMemcpyDeviceToHost);

		hipFree(d_C);
		hipFree(mat_A);
		hipFree(mat_B);

	
	//���GPU, CPU ����Ľ���Ƿ���ͬ
	printf("Checking answer...\n");
	if (!CheckAnswer(E, D, height_A * width_B))
		printf("The answer1 is wrong!");
	else printf("The answer1 is right!");

	if (!CheckAnswer(E, F, height_A * width_B))
		printf("The answer2 is wrong!");
	else printf("The answer2 is right!");

	//if (!CheckAnswer(C, F, height_A * width_B))
	//	printf("The answer3 is wrong!");
	//else printf("The answer3 is right!");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	getchar();
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int WA, unsigned int HA, unsigned int WB, unsigned int HB, Type mode)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, HA * WB * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, HA * WA * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, HB * WB * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, HA * WA * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, HB * WB * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//Ϊÿһ��C[i][j]����һ���߳̽��м���
	int block_size = 16;

	dim3 Threads(block_size, block_size);
	dim3 Blocks(WB / block_size, HA / block_size);

	// Launch a kernel on the GPU with one thread for each element.
	if (mode == Mode1)
	{
		hipEvent_t start1, stop1;
		hipEventCreate(&start1);
		hipEventCreate(&stop1);
		hipEventRecord(start1, 0);

		MatrixMulGPU_1 << <Blocks, Threads >> >(dev_c, dev_a, dev_b, WA, WB);


		hipEventRecord(stop1, 0);
		hipEventSynchronize(stop1);

		float elaspsedTime1;
		hipEventElapsedTime(&elaspsedTime1, start1, stop1);
		printf("GPU time : %3.1fms \n", elaspsedTime1);
	}

	if (mode == Mode2)
	{
		hipEvent_t start2, stop2;
		hipEventCreate(&start2);
		hipEventCreate(&stop2);
		hipEventRecord(start2, 0);

		MatrixMulGPU_2<16> << <Blocks, Threads >> >(dev_c, dev_a, dev_b, WA, WB);

		hipEventRecord(stop2, 0);
		hipEventSynchronize(stop2);

		float elaspsedTime2;
		hipEventElapsedTime(&elaspsedTime2, start2, stop2);
		printf("GPU with Shared Memory time : %3.1fms \n", elaspsedTime2);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	//cudaStatus = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//	goto Error;
	//}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, HA * WB * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
