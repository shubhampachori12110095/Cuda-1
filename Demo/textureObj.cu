/*
	Practise the Texture memory to help me understand the cuda

	DateTime 2016.05.10

*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <malloc.h>
#include <Windows.h>

//Texture Object
//struct hipTextureDesc
//{
//	enum hipTextureAddressMode addressMode[3];
//	enum hipTextureFilterMode	filterMode;
//	enum hipTextureReadMode	readMode;
//
//	int							sRGB;
//	int							normalizedCoords;
//	unsigned int				maxAnisotropy;
//	enum hipTextureFilterMode	mipmapFilterMode;
//	float						minMipmapLevelClamp;
//	float						maxMipmapLevelClamp;
//};

int main()
{
	const int width = 1024;
	const int height = 1024;

	//������CPU�ϵ������������
	float *B = (float *)malloc(sizeof(float) * height * width);
	float *A = (float *)malloc(sizeof(float) * height * width);

	memset(A, 0.0, sizeof(float) * height * width);
	memset(B, 0.0, sizeof(float) * height * width);
	//�����Դ�
	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, width, height); //����cuArray
	
	//��host data����srcָ��Ĵ洢���򿽱����ݵ�hipArray��
	hipMemcpyToArray(cuArray, 0, 0, A, sizeof(float) * 1024 * 1024, hipMemcpyHostToDevice);

	//ָ������
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	//ָ������object�Ĳ���
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]	=	hipAddressModeWrap;
	texDesc.addressMode[1]	=	hipAddressModeWrap;
	texDesc.filterMode		=	hipFilterModeLinear;
	texDesc.readMode		=	hipReadModeElementType;
	texDesc.normalizedCoords =	1;

	//Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	//Allocate the output maxtrix
	float* C;
	hipMalloc(&C, width * height * sizeof(float));

	//Invoke kernel
	dim3 dimBlock;
	dim3 dimGrid;


	//Destory texture object
	hipDestroyTextureObject(texObj);

	//Free the device memory;
	hipFreeArray(cuArray);
	hipFree(C);

	free(A);
	free(B);

    return 0;
}
